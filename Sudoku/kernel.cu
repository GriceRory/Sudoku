

#include "hip/hip_runtime.h"

//#include "cuNet.h"
#include <stdio.h>

struct board {
    int* contents;
    int* guesses;
    int subBoardDimentions;
};

void set_element(board b, int row, int col, int val);
int get_element(board b, int row, int col);
int validBoard(board b);
int validSubBoard(board b, int sub);
int validCol(board b, int col);
int validCol(board b, int row);

int main(){

    return 0;
}




void set_element(board b, int row, int col, int val) {
    b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col] = val;
}
int get_element(board b, int row, int col) {
    return b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col];
}

//rules
int validSubBoard(board b, int sub) {
    int rowStart = (int)(sub / b.subBoardDimentions);
    int colStart = (int)((rowStart - (sub / b.subBoardDimentions)) * b.subBoardDimentions);
    int rowEnd = rowStart + b.subBoardDimentions;
    int colEnd = colStart + b.subBoardDimentions;
    for (int row = rowStart; row < rowEnd; ++row) {
        for (int col = colStart; col < colEnd; ++col) {
            if (!get_element(b, row, col)) {
                continue;
            }
            for (int rowAhead = row + 1; rowAhead < rowEnd; ++rowAhead) {
                for (int colAhead = col + 1; colAhead < colEnd; ++colAhead) {
                    if (get_element(b, row, col) == get_element(b, rowAhead, colAhead)) {
                        return 0;
                    }
                }
            }

        }
    }


    return 1;
}
int validCol(board b, int col) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int row = 0; row < boardSize; ++row) {
        if (get_element(b, row, col) == 0) { continue; }
        for (int rowAhead = row + 1; rowAhead < boardSize; ++rowAhead) {
            if (get_element(b, row, col) == get_element(b, rowAhead, col)) { return 0; }
        }
    }
    return 1;
}
int validRow(board b, int row) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int col = 0; col < boardSize; ++col) {
        if (get_element(b, row, col) == 0) { continue; }
        for (int colAhead = col + 1; colAhead < boardSize; ++colAhead) {
            if (get_element(b, row, col) == get_element(b, row, colAhead)) { return 0; }
        }
    }
    return 1;
}
//rules end

int validBoard(board b) {
    //checking each subboard is valid
    int boardDim = b.subBoardDimentions * b.subBoardDimentions;

    for (int subBoard = 0; subBoard < boardDim; ++subBoard) {
        if (!validSubBoard(b, subBoard)) {
            return 0;
        }
    }

    //checking each row is valid
    for (int row = 0; row < boardDim; ++row) {
        validRow(b, row);
    }

    //checking each column is valid
    for (int col = 0; col < boardDim; ++col) {
        validRow(b, col);
    }


    return 1;
}