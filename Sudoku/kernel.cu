
#include "hip/hip_runtime.h"


#include <stdio.h>

struct board {
    int* contents;
    int subBoardDimentions;
};

void set_element(board b, int row, int col, int val);
int get_element(board b, int row, int col);
int validBoard(board b);
int validSubBoard(board b, int sub);

int main()
{

    return 0;
}

//+-+-+-++-+-+-+-+-+
//|5| | || | | | | |
//+-+-+-++-+-+-+-+-+
//|5| | || | | | | |
//+-+-+-++-+-+-+-+-+
//|5| | || | | | | |
//+-+-+-++-+-+-+-+-+
//+-+-+-++-+-+-+-+-+


void set_element(board b, int row, int col, int val) {
    b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col] = val;
}
int get_element(board b, int row, int col) {
    return b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col];
}

//rules
int validSubBoard(board b, int sub) {
    int rowStart = (int)(sub / b.subBoardDimentions);
    int colStart = (int)((rowStart - (sub / b.subBoardDimentions)) * b.subBoardDimentions);
    int rowEnd = rowStart + b.subBoardDimentions;
    int colEnd = colStart + b.subBoardDimentions;
    for (int row = rowStart; row < rowEnd; ++row) {
        for (int col = colStart; col < colEnd; ++col) {
            if (!get_element(b, row, col)) {
                continue;
            }
            for (int rowAhead = row + 1; rowAhead < rowEnd; ++rowAhead) {
                for (int colAhead = col + 1; colAhead < colEnd; ++colAhead) {
                    if (get_element(b, row, col) == get_element(b, rowAhead, colAhead)) {
                        return 0;
                    }
                }
            }

        }
    }


    return 1;
}
int validCol(board b, int col) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int row = 0; row < boardSize; ++row) {
        for (int rowAhead = row + 1; rowAhead < boardSize; ++rowAhead) {
            if (get_element(b, row, col) == get_element(b, rowAhead, col)) { return 0; }
        }
    }
    return 1;
}
int validRow(board b, int row) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int col = 0; col < boardSize; ++col) {
        for (int colAhead = col + 1; colAhead < boardSize; ++colAhead) {
            if (get_element(b, row, col) == get_element(b, row, colAhead)) { return 0; }
        }
    }
    return 1;
}
//rules end

int validBoard(board b) {
    //checking each subboard is valid
    for (int subBoard = 0; subBoard < b.subBoardDimentions * b.subBoardDimentions; ++subBoard) {
        if (!validSubBoard(b, subBoard)) {
            return 0;
        }
    }
    //checking each row is valid



    //checking each column is valid



    return 1;
}