

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "../../GriceRory/cuNet/cuNet/cuNet/cuNet.h"

struct board {
    int* contents;
    int* guesses;
    int subBoardDimentions;
};

int get_guess(board b, int row, int col);
void set_guess(board b, int row, int col, int val);
void set_element(board b, int row, int col, int val);
int get_element(board b, int row, int col);

int validBoard(board b);
int validSubBoard(board b, int sub);
int validCol(board b, int col);
int validCol(board b, int row);

void guess_element(board b, int row, int col, network net);


int main(){

    return 0;
}

//AI
void guess_element(board b, int row, int col, network net) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    vector *guesses = build_vector(boardSize);
    vector* input = build_vector(boardSize*boardSize);
    run_network(net, *input, guesses);
}
//AI ends



//memory manipulation
void set_guess(board b, int row, int col, int val) {
    b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col] = val;
}
int get_guess(board b, int row, int col) {
    return b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col];
}
void set_element(board b, int row, int col, int val) {
    b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col] = val;
}
int get_element(board b, int row, int col) {
    return b.contents[row * b.subBoardDimentions * b.subBoardDimentions + col];
}
//memory manipulation ends

//rules
int validSubBoard(board b, int sub) {
    int rowStart = (int)(sub / b.subBoardDimentions);
    int colStart = (int)((rowStart - (sub / b.subBoardDimentions)) * b.subBoardDimentions);
    int rowEnd = rowStart + b.subBoardDimentions;
    int colEnd = colStart + b.subBoardDimentions;
    for (int row = rowStart; row < rowEnd; ++row) {
        for (int col = colStart; col < colEnd; ++col) {
            if (!get_element(b, row, col)) {
                continue;
            }
            for (int rowAhead = row + 1; rowAhead < rowEnd; ++rowAhead) {
                for (int colAhead = col + 1; colAhead < colEnd; ++colAhead) {
                    if (get_element(b, row, col) == get_element(b, rowAhead, colAhead)) {
                        return 0;
                    }
                }
            }

        }
    }


    return 1;
}
int validCol(board b, int col) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int row = 0; row < boardSize; ++row) {
        if (get_element(b, row, col) == 0) { continue; }
        for (int rowAhead = row + 1; rowAhead < boardSize; ++rowAhead) {
            if (get_element(b, row, col) == get_element(b, rowAhead, col)) { return 0; }
        }
    }
    return 1;
}
int validRow(board b, int row) {
    int boardSize = b.subBoardDimentions * b.subBoardDimentions;
    for (int col = 0; col < boardSize; ++col) {
        if (get_element(b, row, col) == 0) { continue; }
        for (int colAhead = col + 1; colAhead < boardSize; ++colAhead) {
            if (get_element(b, row, col) == get_element(b, row, colAhead)) { return 0; }
        }
    }
    return 1;
}
//rules end

int validBoard(board b) {
    //checking each subboard is valid
    int boardDim = b.subBoardDimentions * b.subBoardDimentions;

    for (int subBoard = 0; subBoard < boardDim; ++subBoard) {
        if (!validSubBoard(b, subBoard)) {
            return 0;
        }
    }

    //checking each row is valid
    for (int row = 0; row < boardDim; ++row) {
        validRow(b, row);
    }

    //checking each column is valid
    for (int col = 0; col < boardDim; ++col) {
        validRow(b, col);
    }


    return 1;
}